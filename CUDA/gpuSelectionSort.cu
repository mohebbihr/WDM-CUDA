#include "hip/hip_runtime.h"
/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__global__ void selection_sort(double *data, double *index, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        float min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            float val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
			// swap index values
			double tmp = index[i];
			index[i] = index[min_idx];
			index[min_idx] = tmp;
        }
    }
}


////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
/*
__global__ void gpuQSort(double *data, double *index, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data,index, left, right);
        return;
    }

    double *lptr = data+left;
    double *rptr = data+right;
    double  pivot = data[(left+right)/2];

    double *lptr_index = index+left;
    double *rptr_index = index+right;
	
    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        double lval = *lptr;
        double rval = *rptr;

	double lval_index = *lptr_index;
        double rval_index = *rptr_index;
		
        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
	    // index array
	    lptr_index ++;
	    lval_index = *lptr_index;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
	    // index array
	    rptr_index --;
	    rval_index = *rptr_index;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
	    // index array
	    *lptr_index++ = rval_index;
            *rptr_index-- = lval_index;			
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        gpuQSort<<< 1, 1, 0, s >>>(data, index, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        gpuQSort<<< 1, 1, 0, s1 >>>(data, index, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}
*/
