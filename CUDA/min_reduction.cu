
#include <hip/hip_runtime.h>
#include <limits.h>

#define NumThread 128
#define NumBlock 32

__global__ void min_reduce(int* In, int* Out, int * OutIdx, int n){
  __shared__ int sdata[NumThread];
  __shared__ int sIdxdata[NumThread];
  unsigned int i = blockIdx.x * NumThread + threadIdx.x;
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = NumBlock * NumThread;
  int myMin = INT_MAX;
  int myMinIdx = -1;

  while (i < n){
    if(In[i] < myMin){
    	myMin = In[i];
	myMinIdx = i;
    }
    i += gridSize;
  }
  sdata[tid] = myMin;
  sIdxdata[tid] = myMinIdx;
  __syncthreads();

  if (NumThread >= 1024){
    if (tid < 512)
    if(sdata[tid] > sdata[tid + 512] ){ sdata[tid] = sdata[tid + 512]; sIdxdata[tid] = sIdxdata[tid + 512]; }
    __syncthreads();
  }
  if (NumThread >= 512){
    if(sdata[tid] > sdata[tid + 256] ){ sdata[tid] = sdata[tid + 256]; sIdxdata[tid] = sIdxdata[tid + 256]; }
    __syncthreads();
  }
  if (NumThread >= 256){
    if(sdata[tid] > sdata[tid + 128] && sdata[tid + 128] !=0){ sdata[tid] =  sdata[tid + 128]; sIdxdata[tid] = sIdxdata[tid + 128]; }
    __syncthreads();
  }
  if (NumThread >= 128){
    if (tid < 64)
    if(sdata[tid] > sdata[tid + 64] ){ sdata[tid] =    sdata[tid + 64]; sIdxdata[tid] = sIdxdata[tid + 64]; }
    __syncthreads();
  }
  //the following practice is deprecated
   if (tid < 32){
    volatile int *smem = sdata;
    volatile int *sidx = sIdxdata;

    if (NumThread >= 64) if(smem[tid] > smem[tid + 32]){ smem[tid] =  smem[tid+32]; sidx[tid] =  sidx[tid+32]; }
    if (NumThread >= 32) if(smem[tid] > smem[tid + 16]){ smem[tid] =  smem[tid+16]; sidx[tid] =  sidx[tid+16]; }
    if (NumThread >= 16) if(smem[tid] > smem[tid + 8]){ smem[tid] =  smem[tid+8]; sidx[tid] =  sidx[tid+8]; }
    if (NumThread >= 8) if(smem[tid] > smem[tid + 4]){ smem[tid] =  smem[tid+4]; sidx[tid] =  sidx[tid+4]; }
    if (NumThread >= 4) if(smem[tid] > smem[tid + 2]){ smem[tid] =  smem[tid+2]; sidx[tid] =  sidx[tid+2]; }
    if (NumThread >= 2) if(smem[tid] > smem[tid + 1]){ smem[tid] =  smem[tid+1]; sidx[tid] =  sidx[tid+1]; }
  }
  if (tid == 0){
    if(sdata[0] < sdata[1] ){ Out[blockIdx.x] = sdata[0]; OutIdx[blockIdx.x] = sIdxdata[0];}
    else{ Out[blockIdx.x] = sdata[1]; OutIdx[blockIdx.x] = sIdxdata[1];}
  }
}




