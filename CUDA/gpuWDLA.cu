#include "hip/hip_runtime.h"
#include "min_reduction.cu"

#define max(a,b) ((a) > (b) ? (a) : (b))

// samemat2 => vector = vector(k1+1:end);
__device__ void ComputeWeight(double * weight, double * samemat, double * diffmat, int len, int k1, int k2, int loopIdx, int numRow)
{
        int i;
        double s_t1, s_t2;
	double d_t1, d_t2;

        for(i=0; i<k1/2; i++) weight[loopIdx * len  + i] = 1;
	for(i= (k1 - 1); i< (k1 + k2/2 - 1); i++) weight[loopIdx * len + i] = 1;

        s_t1 = samemat[loopIdx * numRow + k1 - 1] - samemat[loopIdx * numRow + k1/2];
        s_t2 = samemat[loopIdx * numRow + k1 - 1] + samemat[loopIdx * numRow + k1/2];
	d_t1 = diffmat[loopIdx * numRow + k2 - 1] - diffmat[loopIdx * numRow + k2/2];
	d_t2 = diffmat[loopIdx * numRow + k2 - 1] + diffmat[loopIdx * numRow + k2/2];

        for(i=k1/2; i<(k1 -1); i++){
		weight[loopIdx * len + i] = ((samemat[loopIdx * numRow + k1 - 1] - samemat[loopIdx * numRow + i]) / s_t1 ) * (s_t2 / (samemat[loopIdx * numRow + k1 - 1] + samemat[loopIdx * numRow + i]));
        }
	for(i= (k1 + k2/2 - 1) ; i< (k1 + k2 - 1); i++){
		weight[loopIdx * len + i] = ((diffmat[loopIdx * numRow + k2 - 1] - diffmat[loopIdx * numRow + (i - k1 + 1)]) / d_t1 ) * (d_t2 / (diffmat[loopIdx * numRow + k2 - 1 ] + diffmat[loopIdx * numRow + (i - k1 + 1)]));
	}

}

__device__ void WeightLi(double * WLi, double * weight, double * omega, int k1, int k2, double beta, int loopIdx, int numRow, int wli_w, int w_w, int o_w )
{
        int i,j;
        double sumomega = 0.0;

        for(i=0; i< k1; i++) omega[loopIdx * o_w + i] = 1;
        for(i=k1; i< (k1 + k2); i++) omega[loopIdx * o_w + i] = - beta;
        for(i=0; i< (k1 + k2); i++) omega[loopIdx * o_w + i] = omega[loopIdx * o_w + i] * weight[loopIdx * w_w + i];
        for(i=0; i< (k1 + k2); i++) sumomega += omega[loopIdx * o_w + i];
        // create WLi matrix
        //WLi = [sumomega,-omega';-omega,diag(omega)];
        // create diag
        for(i=1; i< (k1 + k2 + 1); i++)
        	for(j=1; j< (k1 + k2 + 1); j++){
                        if(i==j) WLi[loopIdx * wli_w + i * (k1 + k2 + 1) + j] = omega[loopIdx * o_w + i-1];
                        else    WLi[loopIdx * wli_w + i * (k1 + k2 + 1) + j] = 0.0;
                }

        for(i=1; i< (k1 + k2 + 1); i++){
                WLi[loopIdx * wli_w + i] = -omega[loopIdx * o_w + i-1]; // WLi[0][i]
                WLi[loopIdx * wli_w + i * wli_w] = -omega[loopIdx * o_w + i-1]; // WLi[i][0]
        }
	
        WLi[loopIdx * wli_w * wli_w] = sumomega;

}

__device__ void selection_sort(double *data, double *index, int left, int right, int loopIdx, int numRow, int k)
{
    for (int i = left ; i <= right ; ++i)
    {
        double min_val = data[loopIdx * numRow + i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            double val_j = data[loopIdx * numRow + j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[loopIdx * numRow + min_idx] = data[loopIdx * numRow + i];
            data[loopIdx * numRow + i] = min_val;
            // swap index values
            double tmp = index[loopIdx * k + i];
            index[loopIdx * k + i] = index[loopIdx * k + min_idx];
            index[loopIdx * k + min_idx] = tmp;
        }
    }
}


// L is the output 

__global__ void gpu_WDLA(double * out, double * per_idx,double * WLi, double * omega, double beta, double * weight, double * sameid, double * diffid, double * samemat, double * diffmat,double * sidx, double * didx, double * dist, double * idxsame, double * idxdiff, double * gnd, int numRow, int numElemPerThread, int k1, int k2){

  int thread_start_idx = (threadIdx.x + blockIdx.x * blockDim.x) * numElemPerThread;
  int thread_end_idx = thread_start_idx + numElemPerThread;

  // int y, x, k, outIdx;
  int gndIdx, i, j, loopIdx, myclass;
  // double tmp, s = 0.0;
  int idxsame_i =0, idxdiff_i =0;

  for(loopIdx = thread_start_idx; loopIdx< thread_end_idx && loopIdx < numRow; loopIdx++){

  for(gndIdx =0; gndIdx < numRow; gndIdx ++){
	myclass = (int) gnd[loopIdx];
        if(myclass == gnd[gndIdx] && gndIdx != loopIdx ){
		idxsame[loopIdx * numRow + idxsame_i] = gndIdx;
                idxsame_i ++;
        }
        if(gnd[gndIdx] != myclass){
                idxdiff[loopIdx * numRow + idxdiff_i] = gndIdx;
                idxdiff_i ++;
        }
  }
  
  // samemat = Distant(LoopI,idxsame');
  // diffmat = Distant(LoopI,idxdiff');
  if(idxsame_i > 0){
        for(i=0; i< idxsame_i; i++){
		samemat[loopIdx * numRow + i] = dist[loopIdx * numRow + (int)idxsame[loopIdx * numRow + i]];
        }
  }
  if(idxdiff_i > 0){
        for(i=0; i< idxdiff_i; i++){
                diffmat[loopIdx * numRow + i] = dist[loopIdx * numRow + (int)idxdiff[loopIdx * numRow + i]];
        }
  }
  
  // [samedist, sidx] = sort(samemat);
  for(i=0; i< max(idxsame_i, idxdiff_i); i++){
        sidx[loopIdx * (k1 + 1) + i] = i;
        didx[loopIdx * (k2 + 1) + i] = i;
  }

  /*for(i=0; i< k1; i++){
	hipStream_t s1;
    	hipStreamCreateWithFlags( &s1, hipStreamNonBlocking );
    	min_reduce<<<NumBlock,NumThread>>>(samemat,samemat, loopIdx, idxsame_i ); 
    	hipStreamDestroy(s1);
	
  }*/
  selection_sort(samemat, sidx, 0, idxsame_i -1, loopIdx, numRow, k1);
  selection_sort(diffmat, didx, 0, idxdiff_i -1, loopIdx, numRow, k2);

  // sameid = idxsame(sidx);
  for(i=0; i< idxsame_i; i++){
        sameid[loopIdx * (k1 + 1) + i] = idxsame[loopIdx * numRow + (int)sidx[loopIdx * (k1 + 1) + i]];
  }
  for(i=0; i< idxdiff_i; i++){
        diffid[loopIdx * (k2 + 1) + i] = idxdiff[loopIdx * numRow + (int)didx[loopIdx * (k2 + 1) + i]];
  }
  
  // weighted distance
  ComputeWeight(weight, samemat, diffmat, k1 + k2 + 2, k1 + 1, k2 + 1, loopIdx, numRow);
  WeightLi(WLi, weight, omega, (int)k1, (int)k2, beta, loopIdx, numRow, k1 + k2 + 1, k1 + k2 + 2, k1 + k2);

  // per_idx = [LoopI, sameclass', diffclass'];
  per_idx[loopIdx * numRow] = loopIdx;
  for(i=1; i<= k1; i++) per_idx[loopIdx * (k1 + k2 + 1) + i] = sameid[loopIdx * (k1 + 1) + i-1];
  for(i= (k1 + 1); i< (k1 + k2 + 1); i++) per_idx[loopIdx * (k1 + k2 + 1) + i] = diffid[loopIdx * (k2 + 1) + i - (int)(k1 + 1)];

  __syncthreads();
  // L(per_idx,per_idx) = L(per_idx,per_idx) + WLi;
  for(i=0; i< (k1 + k2 + 1); i++){
        for(j=0; j< (k1 + k2 + 1); j++){
                out[ (int)per_idx[i] * numRow + (int)per_idx[j]] = out[ (int)per_idx[i] * numRow + (int)per_idx[j]] + WLi[loopIdx * (k1 + k2 + 1) * (k1 + k2 + 1) + i * (k1 + k2 + 1) + j ];
                __syncthreads();
        }
        __syncthreads();
  }

  __syncthreads();

  } // end of loopIdx for
}

