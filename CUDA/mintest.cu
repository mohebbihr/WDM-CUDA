#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "min_reduction.cu"

int main(int argc, char* argv[]){

  unsigned int length = 20; //1048576;
  int i, Size, min, minIdx;
  int *a, *out, *outIdx, *gpuA, *gpuOut, *gpuOutIdx;

  hipSetDevice(0);
  Size = length * sizeof(int);
  a = (int*)malloc(Size);
  out = (int*)malloc(NumBlock*sizeof(int));
  outIdx = (int*)malloc(NumBlock*sizeof(int));
  for(i=0;i<length;i++) a[i] = (i + 10);

  a[10] = 5;

  hipMalloc((void**)&gpuA,Size);
  hipMalloc((void**)&gpuOut,NumBlock*sizeof(int));
  hipMalloc((void**)&gpuOutIdx,NumBlock*sizeof(int));
  hipMemcpy(gpuA,a,Size,hipMemcpyHostToDevice);
  min_reduce<<<NumBlock,NumThread>>>(gpuA,gpuOut, gpuOutIdx, length);
  hipDeviceSynchronize();
  hipMemcpy(out,gpuOut,NumBlock*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(outIdx,gpuOutIdx,NumBlock*sizeof(int),hipMemcpyDeviceToHost);
 
  //printf("out array \n");
  //for(i=0; i<NumBlock; i++) printf("out[%d] = %d\n", i, out[i]);
  //printf("\n");
  
  min = out[0];
  minIdx = outIdx[0];
  for(i=1;i<NumBlock;i++) if(min > out[i]){ min = out[i]; minIdx = outIdx[i]; }
  printf("min: %d, minIdx: %d \n",min, minIdx);
  return 0;
}
