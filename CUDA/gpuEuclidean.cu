
#include <hip/hip_runtime.h>
#define BlockSize 16
#define Tolerance 0.0001

__global__ void gpuEuclidean(const double *input, int numRow, int numCol, double *output){
  __shared__ float Ys[BlockSize][BlockSize];
  __shared__ float Xs[BlockSize][BlockSize]; 
  int bx = blockIdx.x, by = blockIdx.y;
  int tx = threadIdx.x, ty = threadIdx.y;
  int yBegin = by * BlockSize * numCol;
  int xBegin = bx * BlockSize * numCol;
  int yEnd = yBegin + numCol;
  int y, x, k, outIdx;
  float tmp, s = 0.0;
  
  for(y=yBegin,x=xBegin;y<yEnd;y+=BlockSize,x+=BlockSize){
    Ys[ty][tx] = input[y + ty*numCol + tx];
    Xs[tx][ty] = input[x + ty*numCol + tx];
    __syncthreads();
    for(k=0;k<BlockSize;k++){
      tmp = Ys[ty][k] - Xs[k][tx];
      s += tmp*tmp;
    }
    __syncthreads();
  }
  outIdx = (by*BlockSize + ty) * numRow + bx*BlockSize + tx;
  output[outIdx] = sqrtf(s);
}


__global__ void gpuEuclidean2(const double *inputA,const double *inputB, int numRowA, int numRowB, int numCol, double *output){
  __shared__ float Ys[BlockSize][BlockSize];
  __shared__ float Xs[BlockSize][BlockSize]; 
  int bx = blockIdx.x, by = blockIdx.y;
  int tx = threadIdx.x, ty = threadIdx.y;
  int yBegin = by * BlockSize * numCol;
  int xBegin = bx * BlockSize * numCol;
  int yEnd = yBegin + numCol;
  int y, x, k, outIdx;  
  float tmp, s = 0.0;
  
  for(y=yBegin,x=xBegin;y<=yEnd;y+=BlockSize,x+=BlockSize){
    Ys[ty][tx] = inputA[y + ty*numCol + tx];
    Xs[tx][ty] = inputB[x + ty*numCol + tx];
    __syncthreads();
#pragma unroll
    for(k=0;k<BlockSize;k++){
      tmp = Ys[ty][k] - Xs[k][tx];
      s += tmp*tmp;
    }
    __syncthreads();
  }
  outIdx = (by*BlockSize + ty) * numRowB + bx*BlockSize + tx;
  //output[outIdx] = sqrtf(s);
  output[outIdx] = 10;
}
